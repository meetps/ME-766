#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#define DIM 10
#define BlockSize 32

__global__ void multiplyKernel(long int* A, long int* B, long int* C) 
{
	int WIDTH = DIM;
    long int CValue = 0;

	const int TILE_WIDTH = BlockSize;

    int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

    // Shared memory allocation for storing the values in a tile window
    __shared__ long int As[TILE_WIDTH][TILE_WIDTH];
    __shared__ long int Bs[TILE_WIDTH][TILE_WIDTH];

    for (int k = 0; k < (TILE_WIDTH + WIDTH - 1)/TILE_WIDTH; k++) 
    {
        if (k*TILE_WIDTH + threadIdx.x < WIDTH && Row < WIDTH)
            As[threadIdx.y][threadIdx.x] = A[Row*WIDTH + k*TILE_WIDTH + threadIdx.x];
        else                                                   
         	As[threadIdx.y][threadIdx.x] = 0.0;

        if (k*TILE_WIDTH + threadIdx.y < WIDTH && Col < WIDTH)   
         	Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_WIDTH + threadIdx.y)*WIDTH + Col];
        else                                                   
         	Bs[threadIdx.y][threadIdx.x] = 0.0;
        __syncthreads();

        for (int n = 0; n < TILE_WIDTH; ++n)
        {
        	CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
        } 
        __syncthreads();
    }
    if (Row < WIDTH && Col < WIDTH)
    	C[((blockIdx.y * blockDim.y + threadIdx.y)*WIDTH)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

void matMultCUDA(long int A[][DIM],long int B[][DIM],long int C[][DIM])
{
	// Pointers to arrays
	long int *dev_a, *dev_b, *dev_c;

	// Events to calculate time taken by kernel
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate device memory for the arrays
	hipError_t err = hipMalloc((void**)&dev_a, ((DIM)*(DIM))*sizeof(long int));
	printf("Cuda malloc A:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_b, ((DIM)*(DIM))*sizeof(long int));
	printf("Cuda malloc B:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_c, ((DIM)*(DIM))*sizeof(long int));
	printf("Cuda malloc C:%s \n", hipGetErrorString(err));


	//Copy array A and B on device allocated memory
	err = hipMemcpy(dev_a, A, ((DIM*DIM))*sizeof(long int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device A:%s \n", hipGetErrorString(err));
	err = hipMemcpy(dev_b, B, ((DIM*DIM))*sizeof(long int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device B:%s \n", hipGetErrorString(err));

	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);

	// Record Time 	
	hipEventRecord(start);
	multiplyKernel << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);
	hipEventRecord(stop);

	// Retrieve array C from device memory
	err = hipMemcpy(C, dev_c, ((DIM*DIM))*sizeof(long int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed time is %f ms\n", milliseconds);

	// Free the device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
